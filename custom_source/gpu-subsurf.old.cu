
#include <hip/hip_runtime.h>
#include <iostream>
#include <string>
#include <fstream>
#include <sstream>
#include <vector>
#include <chrono>

using namespace std;

// from https://forums.developer.nvidia.com/t/throughput-test-add-mul-mod-giving-strange-result/32021
// remove when done
#define CUDA_CHECK_RETURN(value) {\
    hipError_t _m_cudaStat = value;\
    if (_m_cudaStat != hipSuccess) {\
        fprintf(stderr, "Error %s at line %d in file %s\n",\
                hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);\
        exit(1);\
    }\
}

struct vec3 {
    double x = 0;
    double y = 0;
    double z = 0;
    bool modified = false;
    int status = 0;
};

struct vertex {
    vec3 position;
    int id;
    int neighboringFaces = 4;
};

struct quadFace {
    int vertexIndex[4];
    vec3 midpoint;
    int midpointVertID;
    int edgeSimplificationMatches = 0;
};

__device__ vertex* objVertices;
__device__ quadFace* objFaces;
__device__ vec3* faceMidpoints;
__device__ quadFace* newFaces;
__device__ vertex* newVertices;

__host__
std::vector<std::string> stringSplit(std::string string, char delimiter) {

    std::vector<std::string> splitString;
    std::string currentString = "";

    for (int i = 0; i < string.length(); i++) {
        if (string[i] == delimiter) {

            splitString.push_back(currentString);
            currentString = "";
        } else {

            currentString += string[i];

            if (i + 1 == string.length()) {
                splitString.push_back(currentString);
            }
        }
    }

    return splitString;
}

__host__
void readObj(std::string path, std::vector<vertex>& vertices, std::vector<quadFace>& faces) {
    
    std::ifstream objFile(path);

    // tell the program to not count new lines
    objFile.unsetf(std::ios_base::skipws);

    std::string objFileLine;

    int dataCount_v = 0;
    int id = 0;

    while (getline(objFile, objFileLine)) {

        std::stringstream ss{objFileLine};
        char objFileLineChar;
        ss >> objFileLineChar;

        std::vector<std::string> lineDataSplitBySpaces = stringSplit(objFileLine, ' ');
        std::string lineType = lineDataSplitBySpaces[0];

        vertex currentVert;

        bool wasVert = false;
        int vertType = 0; // 0 = none, 1 = vert, 2 = texture coordinate, 3 = normal vert

        if (lineType.compare("v") == 0) {
            currentVert.position.x = std::stod(lineDataSplitBySpaces[1]);
            currentVert.position.y = std::stod(lineDataSplitBySpaces[2]);
            currentVert.position.z = std::stod(lineDataSplitBySpaces[3]);
            currentVert.position.modified = true;
            currentVert.id = id;

            wasVert = true;
            vertType = 1;
            dataCount_v++;

        } else if (lineType.compare("f") == 0) {

            quadFace currentFace;

            for (int i = 1; i < lineDataSplitBySpaces.size(); i++) {
                
                std::vector<std::string> lineDataSplitBySlashes = stringSplit(lineDataSplitBySpaces[i], '/');

                // vertex_index, texture_index, normal_index
                currentFace.vertexIndex[i - 1] = std::stod(lineDataSplitBySlashes[0]) - 1;

            }

            faces.push_back(currentFace);
        }

        if (wasVert) {

            if (currentVert.id < dataCount_v) vertices.push_back(currentVert);

            // check for which part of the vert has already been written to since the verts are written before the normals verts
            // if the vert type is 1 (v) and the vert hasnt been modified on the verts array
            if (vertType == 1 && !vertices[(dataCount_v - 1)].position.modified) {

                vertices[(dataCount_v - 1)].position.x = currentVert.position.x;
                vertices[(dataCount_v - 1)].position.y = currentVert.position.y;
                vertices[(dataCount_v - 1)].position.z = currentVert.position.z;
                vertices[(dataCount_v - 1)].position.modified = true;
            }

            id++;
        }
    }

    objFile.close();
}

__global__ 
void catmullClarkFacePointsAndEdges(int facesSize_lcl, int maxVertsAtStart_lcl, int totalNewVertsToAllocate) {

    int i = (blockIdx.x * blockDim.x) + threadIdx.x;

    quadFace currentSubdividedFaces[4];
    
    for (int j = 0; j < 4; j++) currentSubdividedFaces[j].vertexIndex[3] = objFaces[i].midpointVertID; // face point [0] will be the center of the subdivided face

    // vertex ids for the edges

    int vertexIDs[4];
    
    for (int j = 0; j < 4; j++) {

        vec3 edgeAveragePoint;

        vertex edgePoint;

        edgeAveragePoint.x = (objVertices[objFaces[i].vertexIndex[(j + 1) % 4]].position.x + objVertices[objFaces[i].vertexIndex[(j + 0) % 4]].position.x) / 2;
        edgeAveragePoint.y = (objVertices[objFaces[i].vertexIndex[(j + 1) % 4]].position.y + objVertices[objFaces[i].vertexIndex[(j + 0) % 4]].position.y) / 2;
        edgeAveragePoint.z = (objVertices[objFaces[i].vertexIndex[(j + 1) % 4]].position.z + objVertices[objFaces[i].vertexIndex[(j + 0) % 4]].position.z) / 2;

        currentSubdividedFaces[j].vertexIndex[1] = objFaces[i].vertexIndex[(j + 0) % 4];

        // find the averages for the face points

        edgePoint.id = maxVertsAtStart_lcl + (i * 5) + (j + 1);

        vertexIDs[j] = edgePoint.id;

        currentSubdividedFaces[j].vertexIndex[0] = edgePoint.id;
        currentSubdividedFaces[(j + 1) % 4].vertexIndex[2] = edgePoint.id;

        objVertices[vertexIDs[j]].position = edgeAveragePoint;
    }

    for (int j = 0; j < 4; j++) {

        newFaces[(i * 4) + j] = currentSubdividedFaces[j];
    }

    objVertices[objFaces[i].midpointVertID].position = faceMidpoints[i];
}

__global__
void replaceNewVerticesWithOldVertices() {

    newVertices = objVertices;
}

__global__
void averageCornerVertices(int facesSize) {

    int i = (blockIdx.x * blockDim.x) + threadIdx.x;

    for (int j = 0; j < 4; j++) {

        int matchedPoints = 0;

        //vec3 neighboringFaceMidpointsAverage;
        vec3 edgeMidpointsAverage;
        //vec3 finalMidpointAverage;

        for (int k = 0; k < facesSize; k++) {

            for (int l = 0; l < 4; l++) {

                if (
                    newVertices[objFaces[i].vertexIndex[j]].position.x == newVertices[objFaces[k].vertexIndex[l]].position.x &&
                    newVertices[objFaces[i].vertexIndex[j]].position.y == newVertices[objFaces[k].vertexIndex[l]].position.y &&
                    newVertices[objFaces[i].vertexIndex[j]].position.z == newVertices[objFaces[k].vertexIndex[l]].position.z
                ) {

                    edgeMidpointsAverage.x += (objVertices[objFaces[i].vertexIndex[j]].position.x + objVertices[objFaces[k].vertexIndex[(l + 1) % 4]].position.x) / 2;
                    edgeMidpointsAverage.y += (objVertices[objFaces[i].vertexIndex[j]].position.y + objVertices[objFaces[k].vertexIndex[(l + 1) % 4]].position.y) / 2;
                    edgeMidpointsAverage.z += (objVertices[objFaces[i].vertexIndex[j]].position.z + objVertices[objFaces[k].vertexIndex[(l + 1) % 4]].position.z) / 2;

                    matchedPoints++;

                    if (matchedPoints > 3) {

                        k = facesSize;
                        l = 4;
                    }
                }
            }
        }

        // will be re-implemented later
        /*
        for (int k = 0; k < matchedPoints; k++) {

            neighboringFaceMidpointsAverage.x += faceMidpoints[neighboringFaceIDs[k]].x;
            neighboringFaceMidpointsAverage.y += faceMidpoints[neighboringFaceIDs[k]].y;
            neighboringFaceMidpointsAverage.z += faceMidpoints[neighboringFaceIDs[k]].z;
        }

        neighboringFaceMidpointsAverage.x /= matchedPoints;
        neighboringFaceMidpointsAverage.y /= matchedPoints;
        neighboringFaceMidpointsAverage.z /= matchedPoints;
        */

        edgeMidpointsAverage.x /= matchedPoints;
        edgeMidpointsAverage.y /= matchedPoints;
        edgeMidpointsAverage.z /= matchedPoints;

        // will be re-implemented later
        /*
        finalMidpointAverage.x = (neighboringFaceMidpointsAverage.x + edgeMidpointsAverage.x) / 2;
        finalMidpointAverage.y = (neighboringFaceMidpointsAverage.y + edgeMidpointsAverage.y) / 2;
        finalMidpointAverage.z = (neighboringFaceMidpointsAverage.z + edgeMidpointsAverage.z) / 2;
        */

        newVertices[objFaces[i].vertexIndex[j]].position = edgeMidpointsAverage;
    }
}


__global__
void mergeByDistance(int facesSize, int verticesSize) {

    int i = (blockIdx.x * blockDim.x) + threadIdx.x;

    for (int j = 0; j < verticesSize; j++) {

        for (int k = 0; k < 4; k++) {

            if (
                newVertices[newFaces[i].vertexIndex[k]].position.x == newVertices[j].position.x &&
                newVertices[newFaces[i].vertexIndex[k]].position.y == newVertices[j].position.y &&
                newVertices[newFaces[i].vertexIndex[k]].position.z == newVertices[j].position.z
            ) {

                newFaces[i].vertexIndex[k] = j;
                newFaces[i].edgeSimplificationMatches++;

                if (newFaces[i].edgeSimplificationMatches >= 3) return;

                k = 4;
            }
        }
    }
}

__host__
void subdivideMeshFromFile(std::string inputFilePath, std::string outputFilePath, bool mergeMeshByDistance) {

    auto startTime = std::chrono::steady_clock::now();

    std::vector<vertex> vertices;
    std::vector<quadFace> faces;

    const int BLOCK_SIZE = 256;

    std::cout << "[CPU] [readObj] READING MESH FROM " << inputFilePath << endl;
    readObj(inputFilePath, vertices, faces); 
    std::cout << "[CPU] [readObj] FINISHED READING MESH" << endl;

    auto endTime = std::chrono::steady_clock::now();
    std::cout << "[CPU] [main] ELAPSED TIME " << std::to_string(std::chrono::duration_cast<std::chrono::milliseconds>(endTime - startTime).count()) << "MS" << endl;

    int facesSize = faces.size();
    int facesSizeAfterSubdivision = facesSize * 4;
    int verticesSize = vertices.size();
    int totalNewVertsToAllocate = facesSize * 5;

    std::cout << "[CPU] [main] " << std::to_string(facesSize) << " FACES AND " << std::to_string(verticesSize) << " VERTICES READ FROM DISK" << endl;
    std::cout << "[CPU] [main] " << std::to_string(facesSizeAfterSubdivision) << " FACES AND " << std::to_string(verticesSize + totalNewVertsToAllocate) << " VERTICES WILL BE ALLOCATED" << endl;

    vertex* objVertices_tmp = new vertex[verticesSize + totalNewVertsToAllocate]; 
    quadFace* objFaces_tmp = new quadFace[facesSize]; 
    vec3* faceMidpoints_tmp = new vec3[facesSize]; 
    quadFace* newFaces_tmp = new quadFace[facesSize * 4]; 
    vertex* newVertices_tmp = new vertex[verticesSize + totalNewVertsToAllocate]; 
    
    CUDA_CHECK_RETURN(hipMallocManaged((void **)&objVertices_tmp, sizeof(vertex) * (verticesSize + totalNewVertsToAllocate)));
    CUDA_CHECK_RETURN(hipMallocManaged((void **)&objFaces_tmp, sizeof(quadFace) * (facesSize)));
    CUDA_CHECK_RETURN(hipMallocManaged((void **)&faceMidpoints_tmp, sizeof(vec3) * (facesSize)));
    CUDA_CHECK_RETURN(hipMallocManaged((void **)&newFaces_tmp, sizeof(quadFace) * (facesSize * 4)));
    CUDA_CHECK_RETURN(hipMallocManaged((void **)&newVertices_tmp, sizeof(vertex) * (verticesSize + (facesSize * 5))));

    for (int j = 0; j < verticesSize; j++) {

        objVertices_tmp[j] = vertices[j];
    } 

    for (int j = verticesSize; j < verticesSize + totalNewVertsToAllocate; j++) {

        vertex tmp;
        objVertices_tmp[j] = tmp;
    }

    for (int j = 0; j < facesSize; j++) {

        objFaces_tmp[j] = faces[j];
    }

    for (int j = 0; j < verticesSize + totalNewVertsToAllocate; j++) {

        vertex tmp;
        newVertices_tmp[j] = tmp;
    }

    for (int j = 0; j < facesSize; j++) {

        vec3 faceAverageMiddlePoint;

        faceAverageMiddlePoint.x = (
            (objVertices_tmp[objFaces_tmp[j].vertexIndex[0]].position.x) + 
            (objVertices_tmp[objFaces_tmp[j].vertexIndex[1]].position.x) + 
            (objVertices_tmp[objFaces_tmp[j].vertexIndex[2]].position.x) + 
            (objVertices_tmp[objFaces_tmp[j].vertexIndex[3]].position.x)
        ) / 4;

        faceAverageMiddlePoint.y = (
            (objVertices_tmp[objFaces_tmp[j].vertexIndex[0]].position.y) + 
            (objVertices_tmp[objFaces_tmp[j].vertexIndex[1]].position.y) + 
            (objVertices_tmp[objFaces_tmp[j].vertexIndex[2]].position.y) + 
            (objVertices_tmp[objFaces_tmp[j].vertexIndex[3]].position.y)
        ) / 4;

        faceAverageMiddlePoint.z = (
            (objVertices_tmp[objFaces_tmp[j].vertexIndex[0]].position.z) + 
            (objVertices_tmp[objFaces_tmp[j].vertexIndex[1]].position.z) + 
            (objVertices_tmp[objFaces_tmp[j].vertexIndex[2]].position.z) + 
            (objVertices_tmp[objFaces_tmp[j].vertexIndex[3]].position.z)
        ) / 4;

        faceMidpoints_tmp[j] = faceAverageMiddlePoint;
        objFaces_tmp[j].midpointVertID = verticesSize + (j * 5);
    }

    CUDA_CHECK_RETURN(hipMemcpyToSymbol(HIP_SYMBOL(objVertices), &objVertices_tmp, sizeof(objVertices_tmp)));
    CUDA_CHECK_RETURN(hipMemcpyToSymbol(HIP_SYMBOL(objFaces), &objFaces_tmp, sizeof(objFaces_tmp)));
    CUDA_CHECK_RETURN(hipMemcpyToSymbol(HIP_SYMBOL(faceMidpoints), &faceMidpoints_tmp, sizeof(faceMidpoints_tmp)));
    CUDA_CHECK_RETURN(hipMemcpyToSymbol(HIP_SYMBOL(newFaces), &newFaces_tmp, sizeof(newFaces_tmp)));
    CUDA_CHECK_RETURN(hipMemcpyToSymbol(HIP_SYMBOL(newVertices), &newVertices_tmp, sizeof(newVertices_tmp)));

    catmullClarkFacePointsAndEdges<<<(facesSize + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE>>>(facesSize, verticesSize, totalNewVertsToAllocate);
    std::cout << "[GPU] [catmullClarkFacePointsAndEdges] FINISHED CALLING KERNELS" << endl;
    CUDA_CHECK_RETURN(hipDeviceSynchronize());
    std::cout << "[GPU] [catmullClarkFacePointsAndEdges] DONE" << endl;

    replaceNewVerticesWithOldVertices<<<1, 1>>>();
    std::cout << "[GPU] [replaceNewVerticesWithOldVertices] FINISHED CALLING KERNEL" << endl;
    CUDA_CHECK_RETURN(hipDeviceSynchronize());
    std::cout << "[GPU] [replaceNewVerticesWithOldVertices] DONE" << endl;

    averageCornerVertices<<<(facesSize + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE>>>(facesSize);
    std::cout << "[GPU] [averageCornerVertices] FINISHED CALLING KERNELS" << endl;
    CUDA_CHECK_RETURN(hipDeviceSynchronize());
    std::cout << "[GPU] [averageCornerVertices] DONE" << endl;

    if (mergeByDistance) {
        
        mergeByDistance<<<(facesSizeAfterSubdivision + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE>>>(facesSizeAfterSubdivision, verticesSize + totalNewVertsToAllocate);
        std::cout << "[GPU] [mergeByDistance] FINISHED CALLING KERNELS" << endl;
        CUDA_CHECK_RETURN(hipDeviceSynchronize());
        std::cout << "[GPU] [mergeByDistance] DONE" << endl;
    }

    quadFace* newFaces_tmp_returnVal = new quadFace[facesSize * 4]; 
    vertex* newVertices_tmp_returnVal = new vertex[verticesSize + totalNewVertsToAllocate]; 

    std::cout << "[GPU] [cudaMemcpyFromSymbol] COPYING MESH DATA TO HOST" << endl;
    CUDA_CHECK_RETURN(hipMemcpyFromSymbol(&newFaces_tmp_returnVal, HIP_SYMBOL(newFaces), sizeof(newFaces)));
    CUDA_CHECK_RETURN(hipMemcpyFromSymbol(&newVertices_tmp_returnVal, HIP_SYMBOL(newVertices), sizeof(newVertices)));
    CUDA_CHECK_RETURN(hipDeviceSynchronize());
    std::cout << "[GPU] [cudaMemcpyFromSymbol] DONE COPYING MESH DATA TO HOST" << endl;

    endTime = std::chrono::steady_clock::now();
    std::cout << "[CPU] [main] ELAPSED TIME " << std::to_string(std::chrono::duration_cast<std::chrono::milliseconds>(endTime - startTime).count()) << "MS" << endl;

    std::cout << "[CPU] [main] WRITING MESH TO " << outputFilePath << endl;

    std::ofstream objFile;
    objFile.open(outputFilePath, ios::out | ios::trunc);

    objFile << "o EXPERIMENTAL_MESH" << endl;

    for (int i = 0; i < verticesSize + totalNewVertsToAllocate; i++) {
        
        objFile << "v " << std::to_string(newVertices_tmp_returnVal[i].position.x) << " " << std::to_string(newVertices_tmp_returnVal[i].position.y) << " " << std::to_string(newVertices_tmp_returnVal[i].position.z) << endl;
    }

    for (int i = 0; i < facesSizeAfterSubdivision ; i++) {

        objFile << "f ";

        for (int j = 0; j < 4; j++) {

            objFile << std::to_string(newFaces_tmp_returnVal[i].vertexIndex[j] + 1) << " ";
        }

        objFile << endl;
    }

    objFile.close();

    std::cout << "[CPU] [main] DONE WRITING MESH TO DISK" << endl;

    endTime = std::chrono::steady_clock::now();
    std::cout << "[END] PROGRAM TOOK " << std::to_string(std::chrono::duration_cast<std::chrono::milliseconds>(endTime - startTime).count()) << "MS" << endl;
}

int main (void) {

    subdivideMeshFromFile("testMesh.obj", "testMeshOutput.obj", false);

    return 0;
}