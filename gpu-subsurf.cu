
#include <hip/hip_runtime.h>
#include <iostream>
#include <string>
#include <fstream>
#include <sstream>
#include <vector>
#include <chrono>

using namespace std;

// from https://forums.developer.nvidia.com/t/throughput-test-add-mul-mod-giving-strange-result/32021
// remove when done
#define CUDA_CHECK_RETURN(value) {											\
	hipError_t _m_cudaStat = value;										\
	if (_m_cudaStat != hipSuccess) {										\
		fprintf(stderr, "Error %s at line %d in file %s\n",					\
				hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);		\
		exit(1);															\
	} }

struct vec3 {
    double x = 0;
    double y = 0;
    double z = 0;
    bool modified = false;
    int status = 0;
};

struct vec2 {
    double x = 0;
    double y = 0;
};

struct vertex {
    vec3 position;
    vec2 textureCoordinate;
    vec3 normal;
    int id;
    int neighboringFaceIDs[4];
    int neighboringFaces = 4;
    bool alreadyAveraged = false;
};

struct quadFace {
    int vertexIndex[4];
    int edgeVertexIndex[4];
    int textureIndex[4];
    int normalIndex[4];
    vec3 midpoint;
    int midpointVertID;
    int edgeSimplificationMatches = 0;
};

__device__ vertex* objVertices;
__device__ quadFace* objFaces;
__device__ vec3* faceMidpoints;
__device__ quadFace* newFaces;
__device__ vertex* newVertices;

__host__
std::vector<std::string> stringSplit(std::string string, char delimiter) {

    std::vector<std::string> splitString;
    std::string currentString = "";

    for (int i = 0; i < string.length(); i++) {
        if (string[i] == delimiter) {

            splitString.push_back(currentString);
            currentString = "";
        } else {

            currentString += string[i];

            if (i + 1 == string.length()) {
                splitString.push_back(currentString);
            }
        }
    }

    return splitString;
}

__host__
void readObj(std::string path, std::vector<vertex>& vertices, std::vector<quadFace>& faces) {
    
    std::ifstream objFile(path);

    // tell the program to not count new lines
    objFile.unsetf(std::ios_base::skipws);

    std::string objFileLine;

    int dataCount_v = 0;
    int id = 0;

    while (getline(objFile, objFileLine)) {

        std::stringstream ss{objFileLine};
        char objFileLineChar;
        ss >> objFileLineChar;

        std::vector<std::string> lineDataSplitBySpaces = stringSplit(objFileLine, ' ');
        std::string lineType = lineDataSplitBySpaces[0];

        vertex currentVert;

        bool wasVert = false;
        int vertType = 0; // 0 = none, 1 = vert, 2 = texture coordinate, 3 = normal vert

        if (lineType.compare("v") == 0) {
            currentVert.position.x = std::stod(lineDataSplitBySpaces[1]);
            currentVert.position.y = std::stod(lineDataSplitBySpaces[2]);
            currentVert.position.z = std::stod(lineDataSplitBySpaces[3]);
            currentVert.position.modified = true;
            currentVert.id = id;

            wasVert = true;
            vertType = 1;
            dataCount_v++;

        } else if (lineType.compare("f") == 0) {

            quadFace currentFace;

            for (int i = 1; i < lineDataSplitBySpaces.size(); i++) {
                
                std::vector<std::string> lineDataSplitBySlashes = stringSplit(lineDataSplitBySpaces[i], '/');

                // vertex_index, texture_index, normal_index
                currentFace.vertexIndex[i - 1] = std::stod(lineDataSplitBySlashes[0]) - 1;
                currentFace.textureIndex[i - 1] = 0;
                currentFace.normalIndex[i - 1] = 0;

            }

            faces.push_back(currentFace);
        }

        if (wasVert) {

            if (currentVert.id < dataCount_v) vertices.push_back(currentVert);

            // check for which part of the vert has already been written to since the verts are written before the normals verts
            // if the vert type is 1 (v) and the vert hasnt been modified on the verts array
            if (vertType == 1 && !vertices[(dataCount_v - 1)].position.modified) {

                vertices[(dataCount_v - 1)].position.x = currentVert.position.x;
                vertices[(dataCount_v - 1)].position.y = currentVert.position.y;
                vertices[(dataCount_v - 1)].position.z = currentVert.position.z;
                vertices[(dataCount_v - 1)].position.modified = true;
            }

            id++;
        }
    }

    objFile.close();
}

__global__ void catmullClarkFacePointsAndEdges(int facesSize_lcl, int maxVertsAtStart_lcl, int totalNewVertsToAllocate) {

    int i = (blockIdx.x * blockDim.x) + threadIdx.x;

    if (i > facesSize_lcl) return;

    quadFace currentSubdividedFaces[4];
    
    for (int j = 0; j < 4; j++) currentSubdividedFaces[j].vertexIndex[3] = objFaces[i].midpointVertID; // face point [0] will be the center of the subdivided face

    // vertex ids for the edges

    int vertexIDs[4];
    
    for (int j = 0; j < 4; j++) {

        vec3 edgeAveragePoint;

        vertex edgePoint;

        edgeAveragePoint.x = (objVertices[objFaces[i].vertexIndex[(j + 1) % 4]].position.x + objVertices[objFaces[i].vertexIndex[(j + 0) % 4]].position.x) / 2;
        edgeAveragePoint.y = (objVertices[objFaces[i].vertexIndex[(j + 1) % 4]].position.y + objVertices[objFaces[i].vertexIndex[(j + 0) % 4]].position.y) / 2;
        edgeAveragePoint.z = (objVertices[objFaces[i].vertexIndex[(j + 1) % 4]].position.z + objVertices[objFaces[i].vertexIndex[(j + 0) % 4]].position.z) / 2;

        currentSubdividedFaces[j].vertexIndex[1] = objFaces[i].vertexIndex[(j + 0) % 4];

        // find the averages for the face points

        edgePoint.id = maxVertsAtStart_lcl + (i * 5) + (j + 1);

        vertexIDs[j] = edgePoint.id;

        currentSubdividedFaces[j].vertexIndex[0] = edgePoint.id;
        currentSubdividedFaces[(j + 1) % 4].vertexIndex[2] = edgePoint.id;

        objVertices[vertexIDs[j]].position = edgeAveragePoint;
        objFaces[i].edgeVertexIndex[j] = vertexIDs[j];
    }

    for (int j = 0; j < 4; j++) {

        newFaces[(i * 4) + j] = currentSubdividedFaces[j];
    }

    objVertices[objFaces[i].midpointVertID].position = faceMidpoints[i];
}

__global__
void replaceNewVerticesWithOldVertices() {

    newVertices = objVertices;
}

__global__
void averageCornerVertices(int facesSize) {

    int i = (blockIdx.x * blockDim.x) + threadIdx.x;

    if (i > facesSize) return;

    for (int j = 0; j < 4; j++) {

        int matchedPoints = 0;
        int neighboringFaceIDs[4];

        vec3 neighboringFaceMidpointsAverage;
        vec3 edgeMidpointsAverage;
        vec3 finalMidpointAverage;

        for (int k = 0; k < facesSize; k++) {

            for (int l = 0; l < 4; l++) {

                if (objFaces[i].vertexIndex[j] == objFaces[k].vertexIndex[l]) {

                    neighboringFaceIDs[matchedPoints] = k;

                    edgeMidpointsAverage.x += (objVertices[objFaces[i].vertexIndex[j]].position.x + objVertices[objFaces[k].vertexIndex[(l + 1) % 4]].position.x) / 2;
                    edgeMidpointsAverage.y += (objVertices[objFaces[i].vertexIndex[j]].position.y + objVertices[objFaces[k].vertexIndex[(l + 1) % 4]].position.y) / 2;
                    edgeMidpointsAverage.z += (objVertices[objFaces[i].vertexIndex[j]].position.z + objVertices[objFaces[k].vertexIndex[(l + 1) % 4]].position.z) / 2;

                    matchedPoints++;

                    if (matchedPoints > 3) {

                        k = facesSize;
                        l = 4;
                    }
                }
            }
        }

        for (int k = 0; k < matchedPoints; k++) {

            neighboringFaceMidpointsAverage.x += faceMidpoints[neighboringFaceIDs[k]].x;
            neighboringFaceMidpointsAverage.y += faceMidpoints[neighboringFaceIDs[k]].y;
            neighboringFaceMidpointsAverage.z += faceMidpoints[neighboringFaceIDs[k]].z;
        }

        neighboringFaceMidpointsAverage.x /= matchedPoints;
        neighboringFaceMidpointsAverage.y /= matchedPoints;
        neighboringFaceMidpointsAverage.z /= matchedPoints;

        edgeMidpointsAverage.x /= matchedPoints;
        edgeMidpointsAverage.y /= matchedPoints;
        edgeMidpointsAverage.z /= matchedPoints;

        finalMidpointAverage.x = (neighboringFaceMidpointsAverage.x + edgeMidpointsAverage.x) / 2;
        finalMidpointAverage.y = (neighboringFaceMidpointsAverage.y + edgeMidpointsAverage.y) / 2;
        finalMidpointAverage.z = (neighboringFaceMidpointsAverage.z + edgeMidpointsAverage.z) / 2;

        newVertices[objFaces[i].vertexIndex[j]].position = edgeMidpointsAverage;
        for (int k = 0; k < 4; k++) newVertices[objFaces[i].vertexIndex[j]].neighboringFaceIDs[k] = neighboringFaceIDs[k];
        if (matchedPoints < 3) newVertices[objFaces[i].vertexIndex[j]].neighboringFaces = matchedPoints;
    }
}


__global__
void mergeByDistance(int facesSize, int verticesSize) {

    int i = (blockIdx.x * blockDim.x) + threadIdx.x;

    for (int j = 0; j < 4; j++) { // for corner j in face i

        int cornerVertexID = newFaces[i].vertexIndex[j];

        if (cornerVertexID > verticesSize || cornerVertexID < 0) continue;

        for (int k = 0; k < 4; k++) { // for neighboring face k in corner j

            int neighboringFaceID = newVertices[cornerVertexID].neighboringFaceIDs[k];

            if (neighboringFaceID > facesSize || neighboringFaceID < 0) continue;

            for (int l = 0; l < 4; l++) { // for corner l in face k

                int neighboringFaceCornerVertexID = newFaces[neighboringFaceID].vertexIndex[l];

                if (neighboringFaceCornerVertexID > verticesSize || neighboringFaceCornerVertexID < 0) continue;
                
                if (
                    newVertices[cornerVertexID].position.x == newVertices[neighboringFaceCornerVertexID].position.x
                ) {

                    printf("%d\n", i);
                }
            }
        }
    }

    /*if (newFaces[i].edgeSimplificationMatches < 4) {

        for (int j = 0; j < facesSize; j++) {

            if (!(newFaces[j].edgeSimplificationMatches < 4)) continue;

            int matches = 0;

            for (int k = 0; k < 4; k ++) {

                if (!(newFaces[j].edgeSimplificationMatches < 4)) continue;

                for (int l = 0; l < 4; l++) {

                    if (!(newFaces[j].edgeSimplificationMatches < 4)) continue;
                    if (newVertices[newFaces[j].vertexIndex[l]].position.status != 0) continue;

                    if (
                        newVertices[newFaces[i].vertexIndex[k]].position.x == newVertices[newFaces[j].vertexIndex[l]].position.x &&
                        newVertices[newFaces[i].vertexIndex[k]].position.y == newVertices[newFaces[j].vertexIndex[l]].position.y &&
                        newVertices[newFaces[i].vertexIndex[k]].position.z == newVertices[newFaces[j].vertexIndex[l]].position.z &&
                        newFaces[i].vertexIndex[k] != newFaces[j].vertexIndex[l]
                    ) {

                        matches++;

                        newFaces[j].edgeSimplificationMatches++;
                        newFaces[i].edgeSimplificationMatches++;

                        if (!(matches < 1) && newVertices[newFaces[j].vertexIndex[l]].position.status == 0) {

                            newVertices[newFaces[i].vertexIndex[k]].position.status = 1;
                            newFaces[j].vertexIndex[l] = newFaces[i].vertexIndex[k];
                        }
                    }
                }
            }
        }
    }*/
}

int main (void) {

    auto startTime = std::chrono::steady_clock::now();
    
    std::string objPath = "./testMesh.obj";
    std::string objOutputPath = "./testMeshOutput.obj";

    std::vector<vertex> vertices;
    std::vector<quadFace> faces;

    const int blockSize = 256;

    std::cout << "[CPU] READING MESH" << endl;
    readObj(objPath, vertices, faces); 
    std::cout << "[CPU] FINISHED READING MESH" << endl;

    int facesSize = faces.size();
    int facesSizeAfterSubdivision = facesSize * 4;
    int verticesSize = vertices.size();
    int totalNewVertsToAllocate = facesSize * 5;

    vertex* objVertices_tmp = new vertex[verticesSize + totalNewVertsToAllocate]; 
    quadFace* objFaces_tmp = new quadFace[facesSize]; 
    vec3* faceMidpoints_tmp = new vec3[facesSize]; 
    quadFace* newFaces_tmp = new quadFace[facesSize * 4]; 
    vertex* newVertices_tmp = new vertex[verticesSize + totalNewVertsToAllocate]; 
    
    CUDA_CHECK_RETURN(hipMallocManaged((void **)&objVertices_tmp, sizeof(vertex) * (verticesSize + totalNewVertsToAllocate)));
    CUDA_CHECK_RETURN(hipMallocManaged((void **)&objFaces_tmp, sizeof(quadFace) * (facesSize)));
    CUDA_CHECK_RETURN(hipMallocManaged((void **)&faceMidpoints_tmp, sizeof(vec3) * (facesSize)));
    CUDA_CHECK_RETURN(hipMallocManaged((void **)&newFaces_tmp, sizeof(quadFace) * (facesSize * 4)));
    CUDA_CHECK_RETURN(hipMallocManaged((void **)&newVertices_tmp, sizeof(vertex) * (verticesSize + (facesSize * 5))));

    for (int j = 0; j < verticesSize; j++) {

        objVertices_tmp[j] = vertices[j];
    } 

    for (int j = verticesSize; j < verticesSize + totalNewVertsToAllocate; j++) {

        vertex tmp;
        objVertices_tmp[j] = tmp;
    }

    for (int j = 0; j < facesSize; j++) {

        objFaces_tmp[j] = faces[j];
    }

    for (int j = 0; j < verticesSize + totalNewVertsToAllocate; j++) {

        vertex tmp;
        newVertices_tmp[j] = tmp;
    }

    for (int j = 0; j < facesSize; j++) {

        vec3 faceAverageMiddlePoint;

        faceAverageMiddlePoint.x = (
            (objVertices_tmp[objFaces_tmp[j].vertexIndex[0]].position.x) + 
            (objVertices_tmp[objFaces_tmp[j].vertexIndex[1]].position.x) + 
            (objVertices_tmp[objFaces_tmp[j].vertexIndex[2]].position.x) + 
            (objVertices_tmp[objFaces_tmp[j].vertexIndex[3]].position.x)
        ) / 4;

        faceAverageMiddlePoint.y = (
            (objVertices_tmp[objFaces_tmp[j].vertexIndex[0]].position.y) + 
            (objVertices_tmp[objFaces_tmp[j].vertexIndex[1]].position.y) + 
            (objVertices_tmp[objFaces_tmp[j].vertexIndex[2]].position.y) + 
            (objVertices_tmp[objFaces_tmp[j].vertexIndex[3]].position.y)
        ) / 4;

        faceAverageMiddlePoint.z = (
            (objVertices_tmp[objFaces_tmp[j].vertexIndex[0]].position.z) + 
            (objVertices_tmp[objFaces_tmp[j].vertexIndex[1]].position.z) + 
            (objVertices_tmp[objFaces_tmp[j].vertexIndex[2]].position.z) + 
            (objVertices_tmp[objFaces_tmp[j].vertexIndex[3]].position.z)
        ) / 4;

        faceMidpoints_tmp[j] = faceAverageMiddlePoint;
        objFaces_tmp[j].midpointVertID = verticesSize + (j * 5);
    }

    CUDA_CHECK_RETURN(hipMemcpyToSymbol(HIP_SYMBOL(objVertices), &objVertices_tmp, sizeof(objVertices_tmp)));
    CUDA_CHECK_RETURN(hipMemcpyToSymbol(HIP_SYMBOL(objFaces), &objFaces_tmp, sizeof(objFaces_tmp)));
    CUDA_CHECK_RETURN(hipMemcpyToSymbol(HIP_SYMBOL(faceMidpoints), &faceMidpoints_tmp, sizeof(faceMidpoints_tmp)));
    CUDA_CHECK_RETURN(hipMemcpyToSymbol(HIP_SYMBOL(newFaces), &newFaces_tmp, sizeof(newFaces_tmp)));
    CUDA_CHECK_RETURN(hipMemcpyToSymbol(HIP_SYMBOL(newVertices), &newVertices_tmp, sizeof(newVertices_tmp)));

    catmullClarkFacePointsAndEdges<<<(facesSize + blockSize - 1) / blockSize, blockSize>>>(facesSize, verticesSize, totalNewVertsToAllocate);
    std::cout << "[GPU] [catmullClarkFacePointsAndEdges] FINISHED CALLING KERNELS" << endl;
    CUDA_CHECK_RETURN(hipDeviceSynchronize());
    std::cout << "[GPU] [catmullClarkFacePointsAndEdges] DONE" << endl;

    replaceNewVerticesWithOldVertices<<<1, 1>>>();
    std::cout << "[GPU] [replaceNewVerticesWithOldVertices] FINISHED CALLING KERNEL" << endl;
    CUDA_CHECK_RETURN(hipDeviceSynchronize());
    std::cout << "[GPU] [replaceNewVerticesWithOldVertices] DONE" << endl;

    averageCornerVertices<<<(facesSize + blockSize - 1) / blockSize, blockSize>>>(facesSize);
    std::cout << "[GPU] [averageCornerVertices] FINISHED CALLING KERNELS" << endl;
    CUDA_CHECK_RETURN(hipDeviceSynchronize());
    std::cout << "[GPU] [averageCornerVertices] DONE" << endl;

    mergeByDistance<<<(facesSizeAfterSubdivision + blockSize - 1) / blockSize, blockSize>>>(facesSizeAfterSubdivision, verticesSize + totalNewVertsToAllocate);
    std::cout << "[GPU] [mergeByDistance] FINISHED CALLING KERNELS" << endl;
    CUDA_CHECK_RETURN(hipDeviceSynchronize());
    std::cout << "[GPU] [mergeByDistance] DONE" << endl;

    quadFace* newFaces_tmp_returnVal = new quadFace[facesSize * 4]; 
    vertex* newVertices_tmp_returnVal = new vertex[verticesSize + totalNewVertsToAllocate]; 

    std::cout << "[GPU] COPYING MESH DATA TO HOST" << endl;
    CUDA_CHECK_RETURN(hipMemcpyFromSymbol(&newFaces_tmp_returnVal, HIP_SYMBOL(newFaces), sizeof(newFaces)));
    CUDA_CHECK_RETURN(hipMemcpyFromSymbol(&newVertices_tmp_returnVal, HIP_SYMBOL(newVertices), sizeof(newVertices)));
    CUDA_CHECK_RETURN(hipDeviceSynchronize());
    std::cout << "[GPU] DONE COPYING MESH DATA TO HOST" << endl;

    std::cout << "[CPU] WRITING MESH TO DISK" << endl;

    std::ofstream objFile;
    objFile.open(objOutputPath, ios::out | ios::trunc);

    objFile << "o EXPERIMENTAL_MESH" << endl;

    for (int i = 0; i < verticesSize + totalNewVertsToAllocate; i++) {
        
        objFile << "v " << std::to_string(newVertices_tmp_returnVal[i].position.x) << " " << std::to_string(newVertices_tmp_returnVal[i].position.y) << " " << std::to_string(newVertices_tmp_returnVal[i].position.z) << endl;
    }

    for (int i = 0; i < facesSizeAfterSubdivision ; i++) {

        objFile << "f ";

        for (int j = 0; j < 4; j++) {

            objFile << std::to_string(newFaces_tmp_returnVal[i].vertexIndex[j] + 1) << " ";
        }

        objFile << endl;
    }

    objFile.close();

    std::cout << "[CPU] DONE WRITING MESH TO DISK" << endl;

    auto endTime = std::chrono::steady_clock::now();
    std::cout << "[END] PROGRAM TOOK " << std::to_string(std::chrono::duration_cast<std::chrono::milliseconds>(endTime - startTime).count()) << "MS" << endl;

    return 0;
}