#include "hip/hip_runtime.h"
// trying to get the catmull clark subdiv method working in c++ so I can translate it to cuda

#include <iostream>
#include <math.h>
#include <string>
#include <fstream>
#include <iterator>
#include <sstream>
#include <vector>
#include <thread>
#include <mutex>
#include <atomic>
#include <map>

using namespace std;

std::mutex threadingMutex;

struct vec3 {
    double x = 0;
    double y = 0;
    double z = 0;
    bool modified = false;
    int status = 0;
};

struct vec2 {
    double x = 0;
    double y = 0;
};

struct vertex {
    vec3 position;
    vec2 textureCoordinate;
    vec3 normal;
    int id;
    int neighboringFaceIDs[4];
    bool alreadyAveraged = false;
};

struct quadFace {
    int vertexIndex[4];
    int edgeVertexIndex[4];
    int textureIndex[4];
    int normalIndex[4];
    vec3 midpoint;
    int midpointVertID;
    int edgeSimplificationMatches = 0;
};

std::vector<std::string> stringSplit(std::string string, char delimiter) {

    std::vector<std::string> splitString;
    std::string currentString = "";

    for (int i = 0; i < string.length(); i++) {
        if (string[i] == delimiter) {

            splitString.push_back(currentString);
            currentString = "";
        } else {

            currentString += string[i];

            if (i + 1 == string.length()) {
                splitString.push_back(currentString);
            }
        }
    }

    return splitString;
}

// can be gpu accelerated at least a little bit since it requires many lines to be read
// for a simple cube there will be no speedup, but for a mesh with millions of verts it will be faster
// currently only reads verts, faces and edges are todo
__global__
void readObj(std::string path, std::vector<vertex>& vertices, std::vector<quadFace>& faces) {
    
    std::ifstream objFile(path);

    // tell the program to not count new lines
    objFile.unsetf(std::ios_base::skipws);

    std::string objFileLine;

    int dataCount_v = 0;
    int id = 0;

    while (getline(objFile, objFileLine)) {

        std::stringstream ss{objFileLine};
        char objFileLineChar;
        ss >> objFileLineChar;

        std::vector<std::string> lineDataSplitBySpaces = stringSplit(objFileLine, ' ');
        std::string lineType = lineDataSplitBySpaces[0];

        vertex currentVert;

        bool wasVert = false;
        int vertType = 0; // 0 = none, 1 = vert, 2 = texture coordinate, 3 = normal vert

        if (lineType.compare("v") == 0) {
            currentVert.position.x = std::stod(lineDataSplitBySpaces[1]);
            currentVert.position.y = std::stod(lineDataSplitBySpaces[2]);
            currentVert.position.z = std::stod(lineDataSplitBySpaces[3]);
            currentVert.position.modified = true;
            currentVert.id = id;

            wasVert = true;
            vertType = 1;
            dataCount_v++;

        } else if (lineType.compare("f") == 0) {

            quadFace currentFace;

            for (int i = 1; i < lineDataSplitBySpaces.size(); i++) {
                
                std::vector<std::string> lineDataSplitBySlashes = stringSplit(lineDataSplitBySpaces[i], '/');

                // vertex_index, texture_index, normal_index
                currentFace.vertexIndex[i - 1] = std::stod(lineDataSplitBySlashes[0]) - 1;
                currentFace.textureIndex[i - 1] = 0;
                currentFace.normalIndex[i - 1] = 0;

            }

            faces.push_back(currentFace);
        }

        if (wasVert) {

            if (currentVert.id < dataCount_v) vertices.push_back(currentVert);

            // check for which part of the vert has already been written to since the verts are written before the normals verts
            // if the vert type is 1 (v) and the vert hasnt been modified on the verts array
            if (vertType == 1 && !vertices[(dataCount_v - 1)].position.modified) {

                vertices[(dataCount_v - 1)].position.x = currentVert.position.x;
                vertices[(dataCount_v - 1)].position.y = currentVert.position.y;
                vertices[(dataCount_v - 1)].position.z = currentVert.position.z;
                vertices[(dataCount_v - 1)].position.modified = true;
            }

            id++;
        }
    }

    objFile.close();
}

// can not be gpu accelerated - is sequental
void writeObj(std::string path, std::vector<vertex> vertices, std::vector<quadFace> faces) {

    std::ofstream objFile;
    objFile.open(path, ios::out | ios::trunc);

    objFile << "o EXPERIMENTAL_MESH" << endl;

    for (int i = 0; i < vertices.size(); i++) {
        
        objFile << "v " << std::to_string(vertices[i].position.x) << " " << std::to_string(vertices[i].position.y) << " " << std::to_string(vertices[i].position.z) << endl;
    }

    for (int i = 0; i < faces.size(); i++) {

        objFile << "f ";

        for (int j = 0; j < 4; j++) {

            objFile << std::to_string(faces[i].vertexIndex[j] + 1) << " ";
        }

        objFile << endl;
    }

    objFile.close();
}

void getVertById(std::vector<vertex> vertices, int id, vertex& vert) {

   vert = vertices[id];
}

void getMaxVertID(std::vector<vertex> vertices, int& max) {

    max = vertices.size();
}

void catmullClarkFacePointsAndEdges(std::vector<vertex>& vertices, std::vector<quadFace>& faces, std::vector<quadFace>& newFaces, int maxVertID, int i, int& completeThreads, int maxVertsAtStart, std::vector<vec3>& faceMidpoints, std::vector<int>& localFaceMidpointVertIDs) {

    quadFace currentSubdividedFaces[4];
    
    for (int j = 0; j < 4; j++) currentSubdividedFaces[j].vertexIndex[3] = localFaceMidpointVertIDs[i]; // face point [0] will be the center of the subdivided face

    // edge midpoints for this face
    // the mesh will have to be combined into one later on, since this will create duplicate verts

    // vertex ids for the edges

    int vertexIDs[4];

    for (int j = 0; j < 4; j++) {

        vec3 edgeAveragePoint;

        vertex edgePoint;

        int neighboringFaceIDs[4];

        int knownFaceID = i;

        bool faceAverageAlreadyCalculated = false;
        int matchedPoints = 0; // the amount of points per face that have been matched

        // find neighboring face
        // search through all faces to find a face sharing points v1, v2 that exist in both the current face and the searching face
        // exclude the current face from the search, therefore the only other possible face containing both points is the desired face
        // this will be optimized later, ignore the 2323978423 nested loops
        
        for (int k = 0; k < faces.size(); k++) {

            for (int l = 0; l < 4; l++) {

                if (faces[i].vertexIndex[j] == faces[k].vertexIndex[l]) {
                    
                    neighboringFaceIDs[matchedPoints] = k;
                    matchedPoints++;
                }
            }
        }


        edgeAveragePoint.x = (vertices[faces[knownFaceID].vertexIndex[(j + 1) % 4]].position.x + vertices[faces[knownFaceID].vertexIndex[(j + 0) % 4]].position.x) / 2;
        edgeAveragePoint.y = (vertices[faces[knownFaceID].vertexIndex[(j + 1) % 4]].position.y + vertices[faces[knownFaceID].vertexIndex[(j + 0) % 4]].position.y) / 2;
        edgeAveragePoint.z = (vertices[faces[knownFaceID].vertexIndex[(j + 1) % 4]].position.z + vertices[faces[knownFaceID].vertexIndex[(j + 0) % 4]].position.z) / 2;

        currentSubdividedFaces[j].vertexIndex[1] = faces[knownFaceID].vertexIndex[(j + 0) % 4];

        // find the averages for the face points

        edgePoint.id = maxVertsAtStart + (i * 5) + (j + 1);

        vertexIDs[j] = edgePoint.id;

        currentSubdividedFaces[j].vertexIndex[0] = edgePoint.id;
        currentSubdividedFaces[(j + 1) % 4].vertexIndex[2] = edgePoint.id;

        threadingMutex.lock();
        vertices[vertexIDs[j]].position = edgeAveragePoint;
        faces[i].edgeVertexIndex[j] = vertexIDs[j];
        threadingMutex.unlock();
    }

    for (int j = 0; j < 4; j++) {

        threadingMutex.lock();
        newFaces.push_back(currentSubdividedFaces[j]);
        threadingMutex.unlock();
    }

    threadingMutex.lock();
    vertices[localFaceMidpointVertIDs[i]].position = faceMidpoints[i];
    threadingMutex.unlock();

    threadingMutex.lock();
    completeThreads++;
    threadingMutex.unlock();
}

void averageCornerVertices(std::vector<vertex>& vertices, std::vector<vertex>& newVertices, std::vector<quadFace>& faces, int i, int& completeThreads, int maxVertsAtStart, std::vector<vec3>& faceMidpoints, std::vector<int>& localFaceMidpointVertIDs) {

    for (int j = 0; j < 4; j++) {

        int matchedPoints = 0;
        int neighboringFaceIDs[4];

        vec3 neighboringFaceMidpointsAverage;
        vec3 edgeMidpointsAverage;
        vec3 finalMidpointAverage;

        for (int k = 0; k < faces.size(); k++) {

            for (int l = 0; l < 4; l++) {

                if (faces[i].vertexIndex[j] == faces[k].vertexIndex[l]) {

                    neighboringFaceIDs[matchedPoints] = k;

                    edgeMidpointsAverage.x += (vertices[faces[i].vertexIndex[j]].position.x + vertices[faces[k].vertexIndex[(l + 1) % 4]].position.x) / 2;
                    edgeMidpointsAverage.y += (vertices[faces[i].vertexIndex[j]].position.y + vertices[faces[k].vertexIndex[(l + 1) % 4]].position.y) / 2;
                    edgeMidpointsAverage.z += (vertices[faces[i].vertexIndex[j]].position.z + vertices[faces[k].vertexIndex[(l + 1) % 4]].position.z) / 2;

                    matchedPoints++;
                }
            }
        }

        for (int k = 0; k < 4; k++) {

            neighboringFaceMidpointsAverage.x += faceMidpoints[neighboringFaceIDs[k]].x;
            neighboringFaceMidpointsAverage.y += faceMidpoints[neighboringFaceIDs[k]].y;
            neighboringFaceMidpointsAverage.z += faceMidpoints[neighboringFaceIDs[k]].z;
        }

        neighboringFaceMidpointsAverage.x /= matchedPoints;
        neighboringFaceMidpointsAverage.y /= matchedPoints;
        neighboringFaceMidpointsAverage.z /= matchedPoints;

        edgeMidpointsAverage.x /= matchedPoints;
        edgeMidpointsAverage.y /= matchedPoints;
        edgeMidpointsAverage.z /= matchedPoints;

        finalMidpointAverage.x = (neighboringFaceMidpointsAverage.x + edgeMidpointsAverage.x) / 2;
        finalMidpointAverage.y = (neighboringFaceMidpointsAverage.y + edgeMidpointsAverage.y) / 2;
        finalMidpointAverage.z = (neighboringFaceMidpointsAverage.z + edgeMidpointsAverage.z) / 2;

        newVertices[faces[i].vertexIndex[j]].position = edgeMidpointsAverage; // find a way to get the finalMidpointAverage to work properly
    }

    threadingMutex.lock();
    completeThreads++;
    threadingMutex.unlock();
}

void mergeByDistance(std::vector<vertex>& vertices, int i, int& completeThreads, std::vector<quadFace>& faces) {

    if (faces[i].edgeSimplificationMatches < 4) {

        for (int j = 0; j < faces.size(); j++) {

            if (!(faces[j].edgeSimplificationMatches < 4)) continue;

            int matches = 0;

            for (int k = 0; k < 4; k ++) {

                if (!(faces[j].edgeSimplificationMatches < 4)) continue;

                for (int l = 0; l < 4; l++) {

                    if (!(faces[j].edgeSimplificationMatches < 4)) continue;

                    if (
                        vertices[faces[i].vertexIndex[k]].position.x == vertices[faces[j].vertexIndex[l]].position.x &&
                        vertices[faces[i].vertexIndex[k]].position.y == vertices[faces[j].vertexIndex[l]].position.y &&
                        vertices[faces[i].vertexIndex[k]].position.z == vertices[faces[j].vertexIndex[l]].position.z &&
                        faces[i].vertexIndex[k] != faces[j].vertexIndex[l]
                    ) {
                        
                        matches++;

                        threadingMutex.lock();
                        faces[j].edgeSimplificationMatches++;
                        faces[i].edgeSimplificationMatches++;
                        threadingMutex.unlock();

                        if (!(matches < 1) && vertices[faces[j].vertexIndex[l]].position.status == 0) {

                            threadingMutex.lock();
                            vertices[faces[i].vertexIndex[k]].position.status = 1;
                            faces[j].vertexIndex[l] = faces[i].vertexIndex[k];
                            threadingMutex.unlock();
                        }
                    }
                }
            }
        }
    }

    threadingMutex.lock();
    completeThreads++;
    threadingMutex.unlock();
}

// adapted from the instructions at https://en.wikipedia.org/wiki/Catmull%E2%80%93Clark_subdivision_surface
// should be gpu accelerated
//__global__
void catmullClarkSubdiv(std::vector<vertex>& vertices, std::vector<quadFace>& faces, const int MAX_CORES, int maxVertsAtStart) {

    const int originalMaxVertID = maxVertsAtStart; // for finding the original non-interpolated verts

    // face points and edge points

    int completeThreads = 0;
    int threadCountOverrunHalts = 0; // the amount of times the program has to stop spawning new threads to wait for the old ones to fall below the MAX_CORES limit

    
    std::atomic<int> workInProgressThreads(0);

    // each thread adds 5 new face points
    // calculate the total new points

    int totalNewVertsToAllocate = faces.size() * 5;

    std::vector<quadFace> newFaces;

    // make new placeholder vertices

    for (int i = 0; i < totalNewVertsToAllocate; i++) {
        
        vertex vert;
        vertices.push_back(vert);
    }

    std::cout << "[CPU] [catmullClarkFacePointsAndEdges()] SPAWNING " << faces.size() << " THREADS" << endl;

    std::vector<vec3> faceMidpoints;
    std::vector<int> localFaceMidpointVertIDs;

    for (int j = 0; j < faces.size(); j++) {

        vec3 faceAverageMiddlePoint;

        faceAverageMiddlePoint.x = (
            (vertices[faces[j].vertexIndex[0]].position.x) + 
            (vertices[faces[j].vertexIndex[1]].position.x) + 
            (vertices[faces[j].vertexIndex[2]].position.x) + 
            (vertices[faces[j].vertexIndex[3]].position.x)
        ) / 4;

        faceAverageMiddlePoint.y = (
            (vertices[faces[j].vertexIndex[0]].position.y) + 
            (vertices[faces[j].vertexIndex[1]].position.y) + 
            (vertices[faces[j].vertexIndex[2]].position.y) + 
            (vertices[faces[j].vertexIndex[3]].position.y)
        ) / 4;

        faceAverageMiddlePoint.z = (
            (vertices[faces[j].vertexIndex[0]].position.z) + 
            (vertices[faces[j].vertexIndex[1]].position.z) + 
            (vertices[faces[j].vertexIndex[2]].position.z) + 
            (vertices[faces[j].vertexIndex[3]].position.z)
        ) / 4;

        faceMidpoints.push_back(faceAverageMiddlePoint);
        localFaceMidpointVertIDs.push_back(maxVertsAtStart + (j * 5) + 0);
    }

    for (int i = 0; i < faces.size(); i++) {

        workInProgressThreads++;
        std::thread(catmullClarkFacePointsAndEdges, std::ref(vertices), std::ref(faces), std::ref(newFaces), originalMaxVertID, i, std::ref(completeThreads), maxVertsAtStart, std::ref(faceMidpoints), std::ref(localFaceMidpointVertIDs)).detach();

        if (i % 100 == 0) {

            std::cout << "[CPU] [catmullClarkFacePointsAndEdges()] " << std::to_string(((float)i / (float)faces.size()) * 100) << "% DONE" << endl;
        }

        while (workInProgressThreads - completeThreads > MAX_CORES) {
            
            threadCountOverrunHalts++;

            if (workInProgressThreads - completeThreads <= MAX_CORES) break;
        }
    };

    std::cout << "[CPU] [catmullClarkFacePointsAndEdges()] THREAD SPAWNING IS DONE" << endl;
    std::cout << "[CPU] [catmullClarkFacePointsAndEdges()] threadCountOverrunHalts=" << std::to_string(threadCountOverrunHalts) << endl;
    std::cout << "[CPU] [catmullClarkFacePointsAndEdges()] WAITING FOR THREADS TO FINISH" << endl;

    while (true) {

        if (workInProgressThreads <= completeThreads) break;
    }

    std::cout << "[CPU] [catmullClarkFacePointsAndEdges()] ALL THREADS ARE DONE" << endl;

    std::cout << "[CPU] [averageCornerVertices()] SPAWNING " << originalMaxVertID << " THREADS" << endl;

    completeThreads = 0;
    workInProgressThreads = 0;
    threadCountOverrunHalts = 0;

    auto newVertices = vertices;

    // neighboring face midpoint gathering
    for (int i = 0; i < faces.size(); i++) {

        workInProgressThreads++;
        std::thread(averageCornerVertices, std::ref(vertices), std::ref(newVertices), std::ref(faces), i, std::ref(completeThreads), maxVertsAtStart, std::ref(faceMidpoints), std::ref(localFaceMidpointVertIDs)).detach();

        if (i % 100 == 0) {

            std::cout << "[CPU] [averageCornerVertices()] " << std::to_string(((float)i / (float)faces.size()) * 100) << "% DONE" << endl;
        }

        while (workInProgressThreads - completeThreads > MAX_CORES) {
            
            threadCountOverrunHalts++;

            if (workInProgressThreads - completeThreads <= MAX_CORES) break;
        }
    }

    std::cout << "[CPU] [averageCornerVertices()] THREAD SPAWNING IS DONE" << endl;
    std::cout << "[CPU] [averageCornerVertices()] threadCountOverrunHalts=" << std::to_string(threadCountOverrunHalts) << endl;
    std::cout << "[CPU] [averageCornerVertices()] WAITING FOR THREADS TO FINISH" << endl;

    while (true) {

        if (workInProgressThreads <= completeThreads) break;
    }

    vertices = newVertices;

    std::cout << "[CPU] [averageCornerVertices()] ALL THREADS ARE DONE" << endl;

    std::cout << "[CPU] [mergeByDistance()] SPAWNING " << originalMaxVertID << " THREADS" << endl;

    completeThreads = 0;
    workInProgressThreads = 0;
    threadCountOverrunHalts = 0;

    faces.clear();
    faces = newFaces;

    // neighboring face midpoint gathering
    for (int i = 0; i < faces.size(); i++) {

        workInProgressThreads++;
        std::thread(mergeByDistance, std::ref(vertices), i, std::ref(completeThreads), std::ref(faces)).detach();

        if (i % (100 * 4) == 0) {

            std::cout << "[CPU] [mergeByDistance()] " << std::to_string(((float)i / (float)faces.size()) * 100) << "% DONE" << endl;
        }

        while (workInProgressThreads - completeThreads > MAX_CORES) {
            
            threadCountOverrunHalts++;

            if (workInProgressThreads - completeThreads <= MAX_CORES) break;
        }
    }

    std::cout << "[CPU] [mergeByDistance()] THREAD SPAWNING IS DONE" << endl;
    std::cout << "[CPU] [mergeByDistance()] threadCountOverrunHalts=" << std::to_string(threadCountOverrunHalts) << endl;
    std::cout << "[CPU] [mergeByDistance()] WAITING FOR THREADS TO FINISH" << endl;

    while (true) {

        if (workInProgressThreads <= completeThreads) break;
    }

    std::cout << "[CPU] [mergeByDistance()] ALL THREADS ARE DONE" << endl;
}

void printVerts(std::vector<vertex> vertices){

    for (int i = 0; i < vertices.size(); i++) {

        std::cout << "[CPU] [" << std::to_string(vertices[i].id) << "] " << "V  : " << std::to_string(vertices[i].position.x) << ", " << std::to_string(vertices[i].position.y) << ", " << std::to_string(vertices[i].position.z) << endl;
        std::cout << "[CPU] [" << std::to_string(vertices[i].id) << "] " << "VN : " << std::to_string(vertices[i].normal.x) << ", " << std::to_string(vertices[i].normal.y) << ", " << std::to_string(vertices[i].normal.z) << endl;
    }
}

void printFaces(std::vector<quadFace> faces, std::vector<vertex> vertices) {

    for (int i = 0; i < faces.size(); i++) {

        // face IDs
        std::cout << "[CPU] Face Vec IDS = ";

        for (int j = 0; j < 4; j++) {

            std::cout << "[V = " << faces[i].vertexIndex[j] << "][VT = " << faces[i].textureIndex[j] << "][VN = " << faces[i].normalIndex[j] << "] ";
        }

        std::cout << endl;

        // face ID values
        std::cout << "[CPU] Vec COORDS   = ";

        for (int j = 0; j < 4; j++) {

            std::cout << "[" << vertices[faces[i].vertexIndex[j]].position.x << ", " << vertices[faces[i].vertexIndex[j]].position.y << ", " << vertices[faces[i].vertexIndex[j]].position.z << "] ";
        }

        std::cout << endl;
    }
}

int main (void) {

    const int MAX_CORES = std::thread::hardware_concurrency() == 0 ? 4 : std::thread::hardware_concurrency();

    std::cout << "[CPU] USING MAX_CORES=" << std::to_string(MAX_CORES) << endl;
    
    std::string objPath = "./testMesh.obj";
    std::string objOutputPath = "./testMeshOutput.obj";
    std::vector<vertex> objVertices;
    std::vector<quadFace> objFaces;

    readObj(objPath, objVertices, objFaces);

    std::string vertCount;
    std::string faceCount;

    vertCount = std::to_string(objVertices.size());
    faceCount = std::to_string(objFaces.size());

    // debugging stuff
    std::cout << "[CPU] FINISHED PARSING \"" << objPath << "\" WITH " << vertCount << " VERTS AND " << faceCount << " FACES" << endl;

    catmullClarkSubdiv(objVertices, objFaces, MAX_CORES, objFaces.size());

    vertCount = std::to_string(objVertices.size());
    faceCount = std::to_string(objFaces.size());

    std::cout << "[CPU] FINISHED SUBDIVIDING \"" << objPath << "\" WITH " << vertCount << " VERTS AND " << faceCount << " FACES" << endl;

    writeObj(objOutputPath, objVertices, objFaces);

    //printVerts(objVertices);
    //printFaces(objFaces, objVertices);

    return 0;
}